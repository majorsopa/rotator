#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "Texture.h"
#include "RotationInfo.h"
#include "texture_getter.cuh"


__global__ void texture_getter(
        Texture * texture_provider,
        std::vector<RotationInfo> * tops_and_bottoms,
        std::vector<RotationInfo> * sides,
        int * start_x,
        int * end_x,
        int * start_y,
        int * end_y,
        int * start_z,
        int * end_z
        ) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < *start_x || x > *end_x || y < *start_y || y > *end_y || z < *start_z || z > *end_z) {
        return;
    }



}

void run_texture_getter(Texture texture_provider, std::vector<RotationInfo> tops_and_bottoms, std::vector<RotationInfo> sides, int start_x, int end_x, int start_y, int end_y, int start_z, int end_z) {
    const int total_x = end_x - start_x;

    texture_getter<<<1, 64>>>(&texture_provider, &tops_and_bottoms, &sides, &start_x, &end_x, &start_y, &end_y, &start_z, &end_z);
}
